#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N (33 * 1024)

__global__ void add( int *a, int *b, int *c ) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main( void ) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the GPU
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

	// fill the arrays 'a' and 'b' on the GPU
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * i;
	}

	// copy the arrays 'a' and 'b' to the GPU
	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyDeviceToHost ) );

	add<<<128, 128>>>( dev_a, dev_b, dev_c );

	// copy the array 'c' back from the GPU to the CPU
	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );
	// verify that the GPU did the work we requested
	bool success = true;
	for (int i = 0; i < N; i++) {
		if ((a[i] + b[i]) != c[i]) {
			printf( "Error: %d + %d != %d\n", a[i], b[i], c[i] );
			success = false;
		}
	}
	if (success)
		printf( "We did it!\n" );

	// free the memory allocated on the GPU
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;
}
